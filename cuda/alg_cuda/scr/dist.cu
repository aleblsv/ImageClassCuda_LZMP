#include "hip/hip_runtime.h"
/**
 ******************************************************************************
 * @file    dist.cu
 * @author  Alexander Belousov
 * @version V1.0.0
 * @date    09/24/2014
 * @brief   Host and Device functions
 ****************************************************************************** 
 */

#include "dist.h"


#define max_val(val1, val2) (val1 > val2 ? val1:val2) /*!< max. val from val1 and val2*/
#define min_val(val1, val2) (val1 < val2 ? val1:val2) /*!< min. val from val1 and val2*/


/**
 *@brief  Compute single distance value from sequence pair LZ-complexity
 *        This function can be called for Host(CPU) or from Device(GPU) 
 *@param  c1: first LZ-complexity value
 *@param  c2: second LZ-complexity value
 *@param  c12: concatenation LZ-complexity value
 *@retval distance value 
 */
float __host__ __device__ dist_calc_val(int c1, int c2, int c12){
	return ((float)(c12 - min_val(c1, c2))/max_val(c1, c2));
}

