#include "hip/hip_runtime.h"
/**
  ******************************************************************************
  * @file    dist.cu
  * @author  alexb
  * @brief
  ******************************************************************************
  * @attention
  *
  * <h2><center>&copy; Copyright (c) 2015, Alexander Belousov and Joel Ratsaby All rights reserved.</center></h2>
  *
  * LICENSE CONDITIONS
  *
  * Redistribution and use in source and binary forms, with or without modification,
  * are permitted provided that the following conditions are met:
  *   1. Redistributions of source code must retain the above copyright notice,
  *      this list of conditions and the following disclaimer.
  *   2. Redistributions in binary form must reproduce the above copyright notice,
  *      this list of conditions and the following disclaimer in the documentation
  *      and/or other materials provided with the distribution.
  *   3. Neither the name of Ariel University nor the names of its contributors
  *      may be used to endorse or promote products derived from this software
  *      without specific prior written permission.
  *
  * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
  * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
  * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
  * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
  * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
  * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
  * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
  * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
  * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
  *
  ******************************************************************************
  */


#include "dist.h"


#define max_val(val1, val2) (val1 > val2 ? val1:val2) /*!< max. val from val1 and val2*/
#define min_val(val1, val2) (val1 < val2 ? val1:val2) /*!< min. val from val1 and val2*/


/**
 *@brief  Compute single distance value from sequence pair LZ-complexity
 *        This function can be called for Host(CPU) or from Device(GPU) 
 *@param  c1: first LZ-complexity value
 *@param  c2: second LZ-complexity value
 *@param  c12: concatenation LZ-complexity value
 *@retval distance value 
 */
float __host__ __device__ dist_calc_val(int c1, int c2, int c12){
	return ((float)(c12 - min_val(c1, c2))/max_val(c1, c2));
}

