/**
 ******************************************************************************
 * @file    my_timer.cu
 * @author  Alexander Belousov
 * @version V1.0.0
 * @date    09/24/2014
 * @brief   Device timer functions
 ****************************************************************************** 
 */

#include <helper_functions.h> // helper utility functions
#include <hip/hip_runtime_api.h>      // helper functions for CUDA error checking and initialization


/**
 *@brief  Private variables
 */
StopWatchInterface *hTm = NULL; /*!< pointer to timer in helper utility    */

/**
 *@brief  Init timer
 *@param  None
 *@retval None
 */
void initTimer(void) {
    sdkCreateTimer(&hTm);
}

/**
 *@brief  Start timer
 *@param  None
 *@retval None
 */
void startTimer(void) {
    sdkResetTimer(&hTm);
    sdkStartTimer(&hTm);
}

/**
 *@brief  Stop timer
 *@param  None
 *@retval None
 */
void stopTimer(void) {
    sdkStopTimer(&hTm);
}

/**
 *@brief  Get timer
 *@param  None
 *@retval timer value in ms
 */
float getTimer(void) {
    return sdkGetTimerValue(&hTm);
}

/**
 *@brief  Delete timer
 *@param  None
 *@retval None
 */
void deleteTimer(void) {
    sdkDeleteTimer(&hTm);
}
